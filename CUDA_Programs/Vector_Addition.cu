#include "hip/hip_runtime.h"
// CUDA Program to Calculate Vector Addition for Two Vectors
// Author: Sandesh Katakam

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

// CHECK ERROR Macro (Reuse this!!)
#define CHECK_ERROR(call) { \
hipError_t err = call; \
if (err != hipSuccess) { \
printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
exit(err); \
} \
}

// compute vector sum C = A+B
// each thread performs one pair-wise addition
// Notes:
// We use i<n in the if condition to faciliate computation of arbitrary length of vectors
// otherwise we compute only mulitples of Thread_numbers(for example 256)
// we stop the comptuation after i reaches the value of the length of the vector
// blockDim, blockIdx, threadIdx are pre-defined variables in the CUDA Language extension
// these take different values for each thread and that's the reason we skip the for loop and just use if statement
// i is the global index calculated uniquely for each thread
// Below this the actual kernel code that is doing the comptuation (in this case vector sum)

__global__ // executed on the device, only callable from the host
void vecAddKernel(float *A, float *B, float *C, int n) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	
	if (i < n) {
		C[i] = A[i] + B[i];
	}
}
// Notes:
// Here the VecAdd is the kernel launching code
// It contains the above VecAddKernel code to invoke the computation 
// After copying necessary variables onto the device
//CHECK_ERROR macro wrapped around hipMalloc gives a descriptive error (easy to debug)
// ceil(n/256.0) can also be computed as (n+256-1/256)
// <<< num_blocks_needed, num_threads_per_block  >>> is specific to CUDA kernels whcih contains thread and block related specifications along with parameters
// Free the Device variables will take away the variable pointers to the available device memory pool

__global__ 
void VecAdd(float* A_h, float* B_h, float* C_h, int n){

    int size = n*sizeof(n);

    float *A_d, float *B_d, float* C_d;

    // Allocate the Variables on the Device (Device Variables)
    CHECK_ERROR(hipMalloc(void**)&A_d, size);
    CHECK_ERROR(hipMalloc(void**)&B_d, size);
    CHECK_ERROR(hipMalloc(void**)&C_d, size);
    
    // Copy Variables from host to Device
    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);
    hipMemcpy(C_d, C_h, size, hipMemcpyHostToDevice);

    // Invoke the CUDA Kernel
    vecAddKernel<<<ceil(n/256.0), 256.0>>>(A_d, B_d, C_d, n);


    // Another way of Invoking the CUDA Kernel with Grids and Block size Specified
    	// Kernel invocation with 256 threads
	// dim3 dimGrid(ceil(n / 256.0),1,1);
	// dim3 dimBlock((256.0),1,1);


    // Copy C_d from device to host C_h
    hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

}

// Host Code Computation of Vector Addition
VecAdd_host(float* A_h, float* B_h, float* C_h, int n){
    for (i = 0; i<n; i++){
        C_h[i] = A_h[i] + B_h[i];
    }
}

int main(){

    // Create host Vectors A_h, B_h and C_h
    A_h = (float*)malloc(sizeof(float)*n);
    B_h = (float*)malloc(sizeof(float)*n);
    C_h = (float*)malloc(sizeof(float)*n);

    // Fill the Host Vectors A_h and B_h with random numbers
    srand(time(NULL));
    for (int i =0; i <= n; i++){
		h_A[i] = ((((float)rand() / (float)(RAND_MAX)) * 100));
		h_B[i] = ((((float)rand() / (float)(RAND_MAX)) * 100));
    }

    // Call VecAdd CUDA Kernel (Invocation from the Main)
    VecAdd(A_h, B_h, C_h, n);

    // Check if the Result of the Vector Sum matches with that of CPU Kernel
    bool valid = true;
    for (int i = 0; i < n ; i++){
        if (A_h[i] + B_h[i] != C_h[i]){
            valid = false;
            break;
        }
    }

    if (!valid){
        printf("The Result is not Correct \n");
    }else{
        printf("The Sum Computed Matches and is Correct \n");
    }

    // Free Host Memory Variables
    free(A_h);
    free(B_h);
    free(C_h);
    return 0;
}